#include "hip/hip_runtime.h"
/*****************************************************************************
 * Copyright (c) 2014-2015 The Parallel Search Team as listed in CREDITS.txt *
 * http://health-tourism.cpe.ku.ac.th/parallelsearch                         *
 *                                                                           *
 * This file is part of ParallelSearch                                       *
 * ParallelSearch is available under multiple licenses.                      *
 * The different licenses are subject to terms and condition as provided     *
 * in the files specifying the license. See "LICENSE.txt" for details        *
 *                                                                           *
 *****************************************************************************
 *                                                                           *
 * ParallelSearch is free software: you can redistribute it and/or modify    *
 * it under the terms of the GNU General Public License as published by      *
 * the Free Software Foundation, either version 3 of the License, or         *
 * (at your option) any later version. See "LICENSE-gpl.txt" for details.    *
 *                                                                           *
 * ParallelSearch is distributed in the hope that it will be useful,         *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of            *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the              *
 * GNU General Public License for more details.                              *
 *                                                                           *
 *****************************************************************************
 *                                                                           *
 * For non-commercial academic use see the license specified in the file     *
 * "LICENSE-academic.txt".                                                   *
 *                                                                           *
 *****************************************************************************
 *                                                                           *
 * If you are interested in other licensing models, including a commercial-  *
 * license, please contact the author at;                                    *
 * Chantana Chantrapornchai <fengcnc@ku.ac.th>                               *
 * Chidchanok Choksuchat <cchoksuchat@hotmail.com>                           *
 *                                                                           *
 *****************************************************************************/
  
///
/// \author Chantana Chantrapornchai <fengcnc@ku.ac.th>
/// \author Chidchanok Choksuchat <cchoksuchat@hotmail.com>
///

/// Version: parallel search on GPU

#define _LFS_LARGEFILE          1
#define _LFS64_LARGEFILE        1
#define _LFS64_STDIO			1
#define _LARGEFILE64_SOURCE    	1

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define BLOCK_SIZE 1014
#define MAX_THREAD_PER_BLOCK 1014
#define MAX 100

long long unsigned total_sub,total_data;
clock_t t_sub1,t_sub2,t_data1,t_data2;
const unsigned long long chunkSize = (1<<20);

unsigned long work_per_thread = 100;
char *pattern_arr[MAX];
int *count_found[MAX];
int total_pattern;
long unsigned total_found;
int TOTAL_THREADS_PER_BLOCK ;
int Rround=0;

__global__ void searchb(char* data, char* pattern, int len_data,int len_substring, bool*pos, unsigned long work_size)
{  

	//For all blocks
	unsigned long k;
	int j,i =blockIdx.x * blockDim.x + threadIdx.x;
	const int numThreads = blockDim.x * gridDim.x;

	for (; i < len_data; i+=numThreads  ) {
	

	if (data[i] == pattern[0]) {
	for ( j=1; i+j < len_data && j<len_substring; j++) {
	if (data[i+j] != pattern[j])     
		break;     
	 }

	if (j==len_substring) {
		 pos[i] =  true;
	}
	else  pos[i] =  false;	   
      }	  
   }
}//end of GPU Kernel

	 FILE* f_b, *pFile = NULL;
	 unsigned long long fileSize = 0;

	 size_t currByte=0;
 	  
	unsigned long long filesize(const char *filename)
	{
	FILE *f = fopen(filename,"rb");  /* open the file in read only */

		if (fseek(f,0,SEEK_END)==0) /* seek was successful for Linux*/
			fileSize = ftell(f);
		fclose(f);
		printf("fileSize = %llu", fileSize);
		return fileSize;
	}

	 int countR=0;

	long unsigned count_total_found(bool *arr, int n)
	{
		 int i;
		 long unsigned c=0;
		 for (i=0; i < n; i++)
			 if (arr[i]) c++;
		 return c;
	}


int main(int argc, char** argv)
{
    printf("start\n");
	int cuda_device = 0; // device ID
        long dposSize =0;
	int mb=0;           // pattern size bit S
	int nb = 0;           // number of ints in the bit data set
	int j,k;


	//start Timer
	hipError_t error;   // capture returned error code
	hipEvent_t start_event, stop_event; // data structures to capture events in GPU
	float time_main_b;
	double total_time_main_b=0.0;
	
	// Sanity checks
	{
	    // check the compute capability of the device A
		int num_devices=0;

		hipGetDeviceCount(&num_devices) ;
		if(0==num_devices)
	    {
	        printf("your system does not have a CUDA capable A device\n");
	        return 1;
	    }
    	if( argc > 1 )
       		cuda_device = atoi( argv[1] );

	    // check if the command-line chosen device ID is within range, exit if not
	    if( cuda_device >= num_devices )
	    {
	        printf("choose device ID between 0 and %d\n", num_devices-1);
	        return 1;
	    }

    	hipSetDevice( cuda_device );

		if ( argc < 4 ) {
      		printf("Usage: StringmatchingGPU <device_number> <data_file_b> <string_pattern1-..99>\n");
      		return -1;
    	}
	} // end of safe checks

	//Cuda Device 
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, cuda_device);
	if( (deviceProp.major == 2) && (deviceProp.minor < 2)){ 
		printf("\n%s does not have compute capability 2.2 or later\n",deviceProp.name);}
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, cuda_device);
	printf(" num SMs %d\n",numSMs);

	//OpenFile
	if ((f_b = fopen(argv[2] , "r")) == NULL ) { 
	printf("Error : read file\n"); 
	return 0; 
	}

	unsigned long long currSize=fileSize;
	long double total_diff2 = 0.0;	
	long double total_time_data = 0.0, total_time_pat =0.0, total_time_pos=0.0;



	while (currSize>chunkSize){
		 currSize=(unsigned long) (currSize-chunkSize);
	 	countR++;
	}
   
	//Substring
	char* subString_b = (char*)malloc( (strlen(argv[3]) + 1) * sizeof(char) );
	strcpy(subString_b, argv[3]);
	 	
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	mb=0;
	for (j=3; j < argc; j++)
	 {
		  pattern_arr[total_pattern] = (char*)malloc( (strlen(argv[j]) + 1) * sizeof(char) ); 
		  count_found[total_pattern] = (int *) malloc( 2*sizeof(int));
		  count_found[total_pattern]=0;
		
		  strcpy(pattern_arr[total_pattern],argv[j]);
		  printf("pattern= %s \n",pattern_arr[total_pattern]);
		   mb= (mb > strlen(pattern_arr[total_pattern])? mb : strlen(pattern_arr[total_pattern]));
	
		  total_pattern++;
	 }

	char* mainString_b;
	char* d_data_b = 0,*data_b;
	bool* pos=false;
	bool* d_pos=false; 
	//Device's text


	// allocate Device's memory for substring
	char* d_substr_b = 0;

	// read in the filename and string pattern to be searched
	int alloc_size =(chunkSize+mb-1)*sizeof(char);
	int countc;
	unsigned int cur_size,my_size;
	char *cur_p,*next_p;
    
        data_b = (char *) malloc((chunkSize+mb-1)*sizeof(char));
	pos = (bool *) malloc((chunkSize+mb-1)*sizeof(bool));
	hipMalloc((void**)&d_pos,(chunkSize+mb-1)*sizeof(bool));//
	if (d_pos == NULL)
		  printf("couldn't allocate d_pos\n");
	          dposSize = dposSize+(long) pos;
		  hipMalloc((void**)&d_data_b, alloc_size) ;//
	
	if (d_data_b == NULL)
		  printf("couldn't allocate d_data_b\n");

	hipMalloc((void**)&d_substr_b, (strlen(subString_b))*sizeof(char));

	size_t cur_free, cur_total;

	printf("\n");

	hipMemGetInfo(&cur_free,&cur_total); 

	printf("%ld KB free of total %ld KB\n",cur_free/1024,cur_total/1024);


	while ( (countc=fread(data_b,sizeof(char),(chunkSize+mb-1),f_b))>0){
	mainString_b = data_b;
	nb = (int) countc/sizeof(char);
	nb= nb-(mb-1);
	printf("size read (byte) %d ", nb); 
		

		TOTAL_THREADS_PER_BLOCK = MAX_THREAD_PER_BLOCK ;

		
		dim3 threadsPerBlocks(TOTAL_THREADS_PER_BLOCK, 1);
		dim3 numBlocks((int)ceil((double)nb/TOTAL_THREADS_PER_BLOCK), 1);

		work_per_thread =(unsigned long) (ceil ((double) BLOCK_SIZE/TOTAL_THREADS_PER_BLOCK));

		printf("numblock %d  thread perblock %d work perThread %ld\n", numBlocks.x, threadsPerBlocks.x, work_per_thread);
		   
		if (work_per_thread <=0) work_per_thread = 1;


		t_data1= clock();
		hipMemcpy(d_data_b, data_b, (nb+(mb-1)), hipMemcpyHostToDevice );
		t_data2= clock();
	        long double diff2 = (((double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
	
	        printf("timeCopyH2D-1 %Lf ms \n",diff2);
                Rround++;
                total_diff2 += diff2;
                total_time_data += diff2;

		// start timer!
		// using Kernel
		for (j=0; j < total_pattern; j++) {
			memset(pos,false,nb);
			hipMemset(d_pos,false,nb);
			t_data1= clock();
			hipMemcpy(d_substr_b, pattern_arr[j], sizeof(char)*(strlen(pattern_arr[j])), hipMemcpyHostToDevice) ;
			
			t_data2= clock();
	                diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
		        printf("timeCopyH2D-2 %Lf ms \n",diff2);

			Rround++;
	                total_diff2 += diff2;
		        total_time_pat += diff2;

			
			 hipEventRecord(start_event, 0);
			
			 searchb <<<32*numSMs,1024>>>(d_data_b, d_substr_b,nb,strlen(pattern_arr[j]),d_pos, work_per_thread  );
			 
			hipEventRecord(stop_event, 0);
			hipEventSynchronize( stop_event );
			//Calculate time
			hipEventElapsedTime( &time_main_b, start_event, stop_event );

			//Getting Error 
			error = hipGetLastError();
			if ( error ) { 	printf("Error caught: %s\n", hipGetErrorString( error ));}
		
			t_data1 =clock();
			hipMemcpy(pos, d_pos, nb, hipMemcpyDeviceToHost) ; // result position
			 t_data2= clock();
		     
			diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
			 printf("timeCopyH2D-3 %Lf ms \n",diff2);
 			 Rround++;
                         total_diff2 += diff2;
                         total_time_pos +=diff2;

			//Print Time
			printf("timeOfMainSearch %f ms ", time_main_b);
			total_time_main_b += time_main_b;
			int t_f =count_total_found(pos,nb);
			printf(" cur_found %d  \n", t_f);
			total_found += t_f;
			// cleanup
			
		}
		// stop timer
	
		if (!feof(f_b)) fseeko(f_b,-((long long)mb-1),SEEK_CUR);
		else break;
             

		}//end while main stream
		
                //Free Substring
		hipFree(d_substr_b);
		free(subString_b);
	
	
		//Free Input

		free(data_b);

		hipFree(d_data_b); 
		hipFree(d_pos); 

		hipEventDestroy( start_event ); 
		hipEventDestroy( stop_event ); 
               
		free(pos);

		for (j=0; j < total_pattern; j++) 
                { free(pattern_arr[j]); 
                  free(count_found[j]); }
		fclose(f_b);

		printf("\nEnd");
		return 0;
}
